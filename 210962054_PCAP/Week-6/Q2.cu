#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include<string.h>

#define N 1024
__global__ void CUDACount(char *A,char* B,int n, int d_count){
int i = threadIdx.x;
int o=n-i;
int start = d_count - (o * (o + 1)) / 2;
     for(int j=0;j<o;j++){
     	B[start+j]=A[j];
     }
   
   }

int main() {
char A[N],B[N];
char *d_A,*d_B;


printf("Enter a string:");
scanf("%s",A);
int c= strlen(A)*(2+(strlen(A)-1))/2;
int size=c*sizeof(char);
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start, 0);
hipMalloc((void**)&d_A, strlen(A)*sizeof(char));

hipMalloc((void**)&d_B, c*sizeof(char));

hipMemcpy(d_A, A, strlen(A)*sizeof(char), hipMemcpyHostToDevice);


hipError_t error =hipGetLastError();
if (error != hipSuccess) {
printf("CUDA Error1: %s\n", hipGetErrorString(error));
}

CUDACount<<<1, strlen(A)>>>(d_A, d_B, strlen(A), c);
error =hipGetLastError();
if (error != hipSuccess) {
printf("CUDA Error2: %s\n", hipGetErrorString(error));
}
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
float elapsedTime;
hipEventElapsedTime(&elapsedTime, start, stop);
hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
B[c] = '\0';
printf("string=%s",B);
hipFree(d_A);

printf("\n");
return 0;
}