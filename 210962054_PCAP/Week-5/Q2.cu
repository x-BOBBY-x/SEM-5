#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Define the size of the vectors
#define N 1024

// CUDA kernel to add two vectors
__global__ void vecAdd(float *a, float *b, float *c) {
  // Get the global thread ID
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  // Make sure we do not go out of bounds
  if (id < N) {
    c[id] = a[id] + b[id];
  }
}

// Main function
int main() {
  // Allocate memory for the vectors on the host
  float *h_a = (float *)malloc(N * sizeof(float));
  float *h_b = (float *)malloc(N * sizeof(float));
  float *h_c = (float *)malloc(N * sizeof(float));

  // Initialize the vectors
  for (int i = 0; i < N; i++) {
    h_a[i] = i;
    h_b[i] = i * 2;
  }

  // Allocate memory for the vectors on the device
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, N * sizeof(float));
  hipMalloc(&d_b, N * sizeof(float));
  hipMalloc(&d_c, N * sizeof(float));

  // Copy the vectors from the host to the device
  hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

  // Calculate the number of blocks
  int numBlocks = N / 256;
  if (N % 256 != 0) {
    numBlocks++;
  }

  // Launch the kernel
  vecAdd<<<numBlocks, 256>>>(d_a, d_b, d_c);

  // Copy the result back from the device to the host
  hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

  // Print the result
  for (int i = 0; i < N; i++) {
    printf("%f\n", h_c[i]);
  }

  // Free the memory on the host
  free(h_a);
  free(h_b);
  free(h_c);

  // Free the memory on the device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}